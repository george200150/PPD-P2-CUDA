﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <chrono> 


__global__ void convoluteGPU(int* pixeliIntrare, int* pixeliIesire, int linii, int coloane, int canaleCuloare) {
	int kernel[5][5] = {
	   { 0,  0, -1,  0,  0},
	   { 0, -1, -2, -1,  0},
	   {-1, -2, 16, -2, -1},
	   { 0, -1, -2, -1,  0},
	   { 0,  0, -1,  0,  0}
	};
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < linii * coloane * canaleCuloare) {
		//out[id] = in[id]+20;

		//apply kernel
		int linie = id / (coloane * canaleCuloare);
		int coloana = (id % (coloane * canaleCuloare)) / canaleCuloare;
		int canalCuloare = id % canaleCuloare;

		int pixel = 0;
		if (linie > 1 && linie < (linii - 2) && coloana > 1 && coloana < (coloane - 2)) {

			//mijloc
			pixel += kernel[2][2] * pixeliIntrare[id];

			//N
			pixel += kernel[1][2] * pixeliIntrare[id - coloane * canaleCuloare];

			//NE
			pixel += kernel[1][3] * pixeliIntrare[id - coloane * canaleCuloare + canaleCuloare];

			//E
			pixel += kernel[2][3] * pixeliIntrare[id + canaleCuloare];

			//SE
			pixel += kernel[3][3] * pixeliIntrare[id + coloane * canaleCuloare + canaleCuloare];

			//S
			pixel += kernel[3][2] * pixeliIntrare[id + coloane * canaleCuloare];

			//SV
			pixel += kernel[3][1] * pixeliIntrare[id + coloane * canaleCuloare - canaleCuloare];

			//V
			pixel += kernel[2][1] * pixeliIntrare[id - canaleCuloare];

			//NV
			pixel += kernel[1][1] * pixeliIntrare[id - coloane * canaleCuloare - canaleCuloare];

			//conturul kernelului
			//N
			pixel += kernel[0][0] * pixeliIntrare[id - 2 * coloane * canaleCuloare - 2 * canaleCuloare];
			pixel += kernel[0][1] * pixeliIntrare[id - 2 * coloane * canaleCuloare - canaleCuloare];
			pixel += kernel[0][2] * pixeliIntrare[id - 2 * coloane * canaleCuloare];
			pixel += kernel[0][3] * pixeliIntrare[id - 2 * coloane * canaleCuloare + canaleCuloare];
			pixel += kernel[0][4] * pixeliIntrare[id - 2 * coloane * canaleCuloare + 2 * canaleCuloare];

			//E
			pixel += kernel[1][4] * pixeliIntrare[id - 1 * coloane * canaleCuloare + 2 * canaleCuloare];
			pixel += kernel[2][4] * pixeliIntrare[id - 0 * coloane * canaleCuloare + 2 * canaleCuloare];
			pixel += kernel[3][4] * pixeliIntrare[id + 1 * coloane * canaleCuloare + 2 * canaleCuloare];

			//S
			pixel += kernel[4][0] * pixeliIntrare[id + 2 * coloane * canaleCuloare - 2 * canaleCuloare];
			pixel += kernel[4][1] * pixeliIntrare[id + 2 * coloane * canaleCuloare - canaleCuloare];
			pixel += kernel[4][2] * pixeliIntrare[id + 2 * coloane * canaleCuloare];
			pixel += kernel[4][3] * pixeliIntrare[id + 2 * coloane * canaleCuloare + canaleCuloare];
			pixel += kernel[4][4] * pixeliIntrare[id + 2 * coloane * canaleCuloare + 2 * canaleCuloare];

			//V
			pixel += kernel[1][0] * pixeliIntrare[id - 1 * coloane * canaleCuloare - 2 * canaleCuloare];
			pixel += kernel[2][0] * pixeliIntrare[id - 0 * coloane * canaleCuloare - 2 * canaleCuloare];
			pixel += kernel[3][0] * pixeliIntrare[id + 1 * coloane * canaleCuloare - 2 * canaleCuloare];

			pixel = pixel / 1;
		}
		else {
			pixel = 0;
		}
		pixeliIesire[id] = pixel;
	}
}

int* mapareMatricePixeliRGBLaVector(int*** imagine, int linii, int coloane, int canaleCuloare) {
	int* vector = (int*)malloc(linii * coloane * canaleCuloare * sizeof(int));
	int id = 0;
	for (int i = 0; i < linii; i++) {
		for (int j = 0; j < coloane; j++) {
			for (int c = 0; c < canaleCuloare; c++) {
				vector[id] = imagine[i][j][c];
				id++;
			}
		}
	}
	return vector;
}

int*** mapareVectorLaMatricePixeliRGB(int* vector, int linii, int coloane, int canaleCuloare) {
	int*** imagine = (int***)malloc(linii * sizeof(int**));
	int id = 0;

	for (int i = 0; i < linii; i++) {
		imagine[i] = (int**)malloc(coloane * sizeof(int*));

		for (int j = 0; j < coloane; j++) {
			imagine[i][j] = (int*)malloc(canaleCuloare * sizeof(int));

			for (int c = 0; c < canaleCuloare; c++) {
				imagine[i][j][c] = vector[id];
				id++;
			}
		}
	}
	return imagine;
}

void aplicareFiltru() {
	//citim matricea de pixeli RGB
	std::ifstream in("pixels.txt");
	int linii, coloane, canaleCuloare;
	in >> linii >> coloane >> canaleCuloare;

	int BLOCK_SIZE = 1000;

	int blockCount = ((linii * coloane * canaleCuloare) / BLOCK_SIZE) + 1;

	//citire in memorie
	int*** matrix = (int***)malloc(linii * sizeof(int**));
	for (int i = 0; i < linii; i++) {
		matrix[i] = (int**)malloc(coloane * sizeof(int*));

		for (int j = 0; j < coloane; j++) {
			int* line = (int*)malloc(canaleCuloare * sizeof(int));

			in >> line[0] >> line[1] >> line[2];

			matrix[i][j] = line;
		}
	}

	int dimensiune = linii * coloane * canaleCuloare;
	//maparea matricei la vector
	int* vector = mapareMatricePixeliRGBLaVector(matrix, linii, coloane, canaleCuloare);
	int* rezultat = (int*)malloc(dimensiune * sizeof(int));

	//copiem vectorul de pixeli in vectorDevice
	int* vectorDevice;
	int* rezultatDevice;
	hipMalloc(&vectorDevice, dimensiune * sizeof(int));
	hipMalloc(&rezultatDevice, dimensiune * sizeof(int));

	hipMemcpy(
		vectorDevice, vector,
		dimensiune * sizeof(int),
		hipMemcpyHostToDevice
	);

	//apelam filtrul convolutional (better: multiplu de 2 ca numar de thread-uri) (test: different block sizes)
	convoluteGPU <<< blockCount, BLOCK_SIZE >>> (vectorDevice, rezultatDevice, linii, coloane, canaleCuloare);

	//copiem rezultatDevice in rezultat
	hipMemcpy(
		rezultat, rezultatDevice,
		dimensiune * sizeof(int),
		hipMemcpyDeviceToHost
	);

	int*** imagine = mapareVectorLaMatricePixeliRGB(rezultat, linii, coloane, canaleCuloare);

	std::ofstream out("pixels.txt");
	out << linii << " " << coloane << " " << canaleCuloare << "\n";
	for (int i = 0; i < linii; i++) {
		for (int j = 0; j < coloane; j++) {
			for (int k = 0; k < canaleCuloare; k++) {
				out << imagine[i][j][k] << " ";
			}
			out << "\n";
		}
	}

	out.close();
}

int main() {
	char* pathFisierIntrare = "python in.py C:/Users/George/source/repos/P2/P2/landscape.png";
	char* pathFisierIesire = "python out.py C:/Users/George/source/repos/P2/P2/landscape1.png";

	system(pathFisierIntrare);			//citim si scriem valoarea pixelilor in pixels.txt

	auto start = std::chrono::steady_clock::now();
	aplicareFiltru();
	auto stop = std::chrono::steady_clock::now();

	system(pathFisierIesire);			//scriem pixelii in imaginea filtrata

	// we cuda've done that
	auto diff = stop - start;
	std::cout << std::chrono::duration <double, std::milli>(diff).count() << " ms" << std::endl;
	
	return 0;
}